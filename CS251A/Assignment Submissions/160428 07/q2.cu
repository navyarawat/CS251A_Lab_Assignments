
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define CUDA_ERROR_EXIT(str) do{\
                                hipError_t err = hipGetLastError();\
                                if( err != hipSuccess){\
                                    printf("Cuda Error: '%s' for %s\n", hipGetErrorString(err), str);\
                                    exit(-1);\
                                }\
                            } while(0);

#define TDIFF(start, end) ((end.tv_sec - start.tv_sec) * 1000000UL + (end.tv_usec - start.tv_usec))

__global__ void xorsum(int* arr, int* num) {
    int arridx = threadIdx.x + blockIdx.x * blockDim.x;
    if (arridx < *num/2)
        arr[arridx] = arr[arridx] ^ arr[arridx + *num/2 + (*num % 2)];
}

__global__ void updatenum(int* num) {
    *num = (*num)/2 + (*num % 2);
}

int debug(int* arr, int num) {
    int sum = 0;
    for (int i = 0; i < num ; i++) {
        sum = sum ^ arr[i];
    }
    return sum;
}

int main(int argc, char **argv) {
    if (argc != 3) {
        printf("Usage: %s <no of elements> <seed>\n",argv[0]);
        exit(-1);
    }
    int N = atoi(argv[1]);
    if (N < 1) {
        printf("Numbers of elements has to be greater than 0\n");
        exit(-1);
    }
    int *hArr;           //host copy of the array
    int size = N * sizeof(int);
    hArr = (int *) malloc(size);
    srand(atoi(argv[2]));
    for (int i = 0; i < N; i++)
        hArr[i] = random();
    /*
    struct timeval seqstart, seqend;
    gettimeofday(&seqstart, NULL);
    printf("%d\n",debug(hArr,N));
    gettimeofday(&seqend, NULL);
    printf("Sequential Processsing time = %ld microsecs\n", TDIFF(seqstart, seqend));
    */

    int *dArr;          //device copy of the array
    int *Num;           //device copy of the number of elements
    hipMalloc(&dArr, size);
    hipMalloc(&Num, sizeof(int));
    CUDA_ERROR_EXIT("hipMalloc");

    // Copy inputs to device
    hipMemcpy(dArr, hArr, size, hipMemcpyHostToDevice);
    hipMemcpy(Num, &N, sizeof(int), hipMemcpyHostToDevice);
    CUDA_ERROR_EXIT("cudaMemcopy");

    //define number of threads and blocks
    struct timeval start, end;
    int threadsPerBlock, blocks;
    gettimeofday(&start, NULL);
    int temp = N;
    printf("Starting...\n");
    while (temp > 1) {
        threadsPerBlock = 1024;
        blocks = temp / 1024 + (temp % 1024 != 0);
        xorsum<<<blocks,threadsPerBlock>>>(dArr,Num);
        updatenum<<<1,1>>>(Num);
        CUDA_ERROR_EXIT("kernel invocation");
        temp = temp/2 + (temp % 2);
    }
    printf("Calculated...\nPrinting result...\n");
    gettimeofday(&end, NULL);

    // Copy result back to host
    int result;
    hipMemcpy(&result, dArr, sizeof(int), hipMemcpyDeviceToHost);
    CUDA_ERROR_EXIT("cudaMemcopy");
    printf("%d\n",result);
    printf("GPU Processsing time = %ld microsecs\n", TDIFF(start, end));

    free(hArr);
    hipFree(dArr);
    return 0;
}
