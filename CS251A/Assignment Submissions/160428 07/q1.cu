
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

#define NUM 10000000        //ten million

#define CUDA_ERROR_EXIT(str) do{\
                                hipError_t err = hipGetLastError();\
                                if( err != hipSuccess){\
                                    printf("Cuda Error: '%s' for %s\n", hipGetErrorString(err), str);\
                                    exit(-1);\
                                }\
                            } while(0);
#define TDIFF(start, end) ((end.tv_sec - start.tv_sec) * 1000000UL + (end.tv_usec - start.tv_usec))

struct num_array{
    double num1;
    double num2;
    double result;
};

__device__ void function(struct num_array *a) {
    double square = a ->num1 * a->num1 +  a->num2 * a->num2  + 2 * a->num1 * a->num2;
    a->result = log(square)/sin(square);
    return;
}

__global__ void calculate(char *mem, int num){
    int t = threadIdx.x + (blockDim.x * threadIdx.y) + (blockDim.x * blockDim.y * blockIdx.x);
    if(t >= num)
        return;
    struct num_array *a = (struct num_array *)(mem + (t * 3 * sizeof(double)));
    function(a);
}

int main(int argc, char **argv) {
    if (argc != 4){
        printf("Usage: %s <no of elements> <rows> <cols>\n",argv[0]);
        exit(-1);
    }
    if (atoi(argv[2]) * atoi(argv[3]) > 1024) {
        printf("<rows> * <cols> should be leq 1024\n");
        exit(-1);
    }
    struct timeval start, end, t_start, t_end;

    unsigned long num = atoi(argv[1]);
    if (num < 1){
        num = NUM;          //NUM defined as a MACRO
    }

    char* ptr = (char*) malloc(num * 3 * sizeof(double));
    char* curr = ptr;
    struct num_array* temp;
    for (int i = 0; i < num; i++){
        temp = (struct num_array*) curr;
        temp->num1 = i + i * 0.1;
        temp->num2 = temp->num1 + 1.0;
        curr += 3 * sizeof(double);
    }           //ptr becomes an undercover array of num_arrays
                //whose result part is yet to be calculated

    char* gpu_mem;      //contents of ptr will be copied over to gpu_mem
    gettimeofday(&t_start, NULL);
    // Allocate GPU memory and copy from CPU --> GPU
    hipMalloc(&gpu_mem, num * 3 * sizeof(double));
    CUDA_ERROR_EXIT("hipMalloc");
    hipMemcpy(gpu_mem, ptr, num * 3 * sizeof(double) , hipMemcpyHostToDevice);
    CUDA_ERROR_EXIT("hipMemcpy");

    gettimeofday(&start, NULL);
    int x = atoi(argv[2]), y = atoi(argv[3]);
    dim3 threadsPerBlock(x, y);
    int numBlocks = num/(x * y) + (num % (x * y) != 0);
    calculate<<<numBlocks, threadsPerBlock>>>(gpu_mem, num);
    CUDA_ERROR_EXIT("kernel invocation");
    gettimeofday(&end, NULL);

    // Copy back results
    hipMemcpy(ptr, gpu_mem, num * 3 * sizeof(double) , hipMemcpyDeviceToHost);
    CUDA_ERROR_EXIT("memcpy");
    gettimeofday(&t_end, NULL);

    printf("Total time = %ld microsecs Processsing = %ld microsecs\n", TDIFF(t_start, t_end), TDIFF(start, end));
    hipFree(gpu_mem);

    //print last element for sanity check
    temp = (struct num_array *) (ptr + (num - 1) * 3 * sizeof(double));
    printf("num1=%f num2=%f result=%f\n", temp->num1, temp->num2, temp->result);
    free(ptr);
}
