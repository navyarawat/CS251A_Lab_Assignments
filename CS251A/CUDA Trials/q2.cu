
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define CUDA_ERROR_EXIT(str) do{\
                                hipError_t err = hipGetLastError();\
                                if( err != hipSuccess){\
                                    printf("Cuda Error: '%s' for %s\n", hipGetErrorString(err), str);\
                                    exit(-1);\
                                }\
                            } while(0);

#define TDIFF(start, end) ((end.tv_sec - start.tv_sec) * 1000000UL + (end.tv_usec - start.tv_usec))

__global__ void xorsum(int* arr, int* num) {
    int arridx = threadIdx.x + blockIdx.x * blockDim.x;
    int totalThreads = blockDim.x;

    while (totalThreads > 1) {
        int halfPoint = (totalThreads >> 1);        //divide by two
        if (threadIdx.x < halfPoint && arridx + halfPoint < *num) {
            arr[arridx] = arr[arridx] ^ arr[arridx + halfPoint];
        }
        __syncthreads();
        totalThreads = halfPoint;
    }
    __syncthreads();
    if (arridx == 0) {
        int temp = blockDim.x;
        while (temp < *num) {
            arr[arridx] = arr[arridx] ^ arr[arridx + temp];
            temp += blockDim.x;
        }
    }
}

int main(int argc, char **argv) {
    if (argc != 3) {
        printf("Usage: %s <no of elements> <seed>\n",argv[0]);
        exit(-1);
    }
    int N = atoi(argv[1]);
    if (N < 1) {
        printf("Numbers of elements has to be greater than 0\n");
        exit(-1);
    }
    int *hArr;           //host copy of the array
    int size = N * sizeof(int);
    hArr = (int *) malloc(size);
    srand(atoi(argv[2]));
    for (int i = 0; i < N; i++)
        hArr[i] = random();

    int *dArr;          //device copy of the array
    int *Num;           //device copy of the number of elements
    hipMalloc(&dArr, size);
    hipMalloc(&Num, sizeof(int));

    // Copy inputs to device
    hipMemcpy(dArr, hArr, size, hipMemcpyHostToDevice);
    hipMemcpy(Num, &N, sizeof(int), hipMemcpyHostToDevice);

    //define number of threads and blocks
    struct timeval start, end;
    int threadsPerBlock = 1024;
    int blocks = N / 1024 + (N % 1024 != 0);
    gettimeofday(&start, NULL);
    xorsum<<<blocks,threadsPerBlock>>>(dArr,Num);
    gettimeofday(&end, NULL);

    // Copy result back to host
    hipMemcpy(hArr, dArr, sizeof(int), hipMemcpyDeviceToHost);
    printf("%d\n",hArr[0]);
    printf("Processsing time = %ld microsecs\n", TDIFF(start, end));

    free(hArr);
    hipFree(dArr);
    return 0;
}
