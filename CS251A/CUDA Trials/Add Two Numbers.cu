#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

void CUDA_ERROR_EXIT(char str[]) {
    hipError_t err = hipGetLastError();
    if( err != hipSuccess){
        char temp[] = hipGetErrorString(err);
        printf("Cuda Error: '%s' for %s\n", temp, str);
        exit(-1);
    }
}

__global__ void add(int *a, int *b, int *c) {
    *c = *a + *b;
}

int main(int argc, char **argv) {
    if (argc != 3) {
        printf("Usage: %s <integer> <integer>",argv[0]);
        exit(-1);
    }
    int ha, hb, hc;           //host copies of variables
    ha = atoi(argv[1]);
    hb = atoi(argv[2]);

    int size = sizeof(int);
    int *da, *db, *dc;          //device copies of variables
    hipMalloc(&da, size);
    hipMalloc(&db, size);
    hipMalloc(&dc, size);

    // Copy inputs to device
    hipMemcpy(da, &ha, size, hipMemcpyHostToDevice);
    hipMemcpy(db, &hb, size, hipMemcpyHostToDevice);

    add<<<1,1>>>(da, db, dc);

    // Copy result back to host
    hipMemcpy(&hc, dc, size, hipMemcpyDeviceToHost);

    hipFree(da); hipFree(db); hipFree(dc);
    return 0;
}
