#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

void CUDA_ERROR_EXIT(char str[]) {
    hipError_t err = hipGetLastError();
    if( err != hipSuccess){
        char temp[] = hipGetErrorString(err);
        printf("Cuda Error: '%s' for %s\n", temp, str);
        exit(-1);
    }
}

__global__ void add(int *a, int *b, int *c) {
    c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

int main(int argc, char **argv) {
    if (argc != 2) {
        printf("Usage: %s <integer> <integer>",argv[0]);
        exit(-1);
    }
    int N = atoi(argv[1]);
    int *ha, *hb, *hc;           //host copies of variables
    int size = N * sizeof(int);
    ha = (int *) malloc(size);
    hb = (int *) malloc(size);
    hc = (int *) malloc(size);
    random_ints(ha, N);
    random_ints(hb, N);

    int *da, *db, *dc;          //device copies of variables
    hipMalloc(&da, size);
    hipMalloc(&db, size);
    hipMalloc(&dc, size);

    // Copy inputs to device
    hipMemcpy(da, ha, size, hipMemcpyHostToDevice);
    hipMemcpy(db, hb, size, hipMemcpyHostToDevice);

    // Launch add() kernel on GPU, with one block running the code on N threads (N < 1024)
    add<<<1,N>>>(d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(&c, dc, size, hipMemcpyDeviceToHost);

    free(a); free(b); free(c);
    hipFree(da); hipFree(db); hipFree(dc);
    return 0;
}
