#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

void CUDA_ERROR_EXIT(char str[]) {
    hipError_t err = hipGetLastError();
    if( err != hipSuccess){
        char temp[] = hipGetErrorString(err);
        printf("Cuda Error: '%s' for %s\n", temp, str);
        exit(-1);
    }
}

__global__ void add(int *a, int *b, int *c, int *N) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if ( i >= *N)
        return;
    c[i] = a[i] + b[i];
}

int main(int argc, char **argv) {
    if (argc != 2) {
        printf("Usage: %s <integer>",argv[0]);
        exit(-1);
    }
    int N = atoi(argv[1]);
    int *ha, *hb, *hc;           //host copies of variables
    int size = N * sizeof(int);
    ha = (int *) malloc(size);
    hb = (int *) malloc(size);
    hc = (int *) malloc(size);
    random_ints(ha, N);
    random_ints(hb, N);

    int *da, *db, *dc;          //device copies of variables
    int *Num;
    hipMalloc(&da, size);
    hipMalloc(&db, size);
    hipMalloc(&dc, size);

    // Copy inputs to device
    hipMemcpy(da, ha, size, hipMemcpyHostToDevice);
    hipMemcpy(db, hb, size, hipMemcpyHostToDevice);
    hipMemcpy(Num, N, sizeof(int), hipMemcpyHostToDevice);

    //define number of threads and blocks
    int threadsPerBlock = 512;
    int blocks = N / 512 + (N % 512 != 0);
    add<<<blocks,threadsPerBlock>>>(d_a, d_b, d_c, Num);

    // Copy result back to host
    hipMemcpy(hc, dc, size, hipMemcpyDeviceToHost);

    free(ha); free(hb); free(hc);
    hipFree(da); hipFree(db); hipFree(dc);
    return 0;
}
