#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

void CUDA_ERROR_EXIT(char str[]) {
    hipError_t err = hipGetLastError();
    if( err != hipSuccess){
        char temp[] = hipGetErrorString(err);
        printf("Cuda Error: '%s' for %s\n", temp, str);
        exit(-1);
    }
}

__global__ void add(int *a, int *b, int *c) {
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

int main(int argc, char **argv) {
    if (argc != 2) {
        printf("Usage: %s <integer> <integer>",argv[0]);
        exit(-1);
    }
    int N = atoi(argv[1]);
    int *ha, *hb, *hc;           //host copies of variables
    int size = N * sizeof(int);
    ha = (int *) malloc(size);
    hb = (int *) malloc(size);
    hc = (int *) malloc(size);
    random_ints(ha, N);
    random_ints(hb, N);

    int *da, *db, *dc;          //device copies of variables
    hipMalloc(&da, size);
    hipMalloc(&db, size);
    hipMalloc(&dc, size);

    // Copy inputs to device
    hipMemcpy(da, ha, size, hipMemcpyHostToDevice);
    hipMemcpy(db, hb, size, hipMemcpyHostToDevice);

    // Launch add() kernel on GPU with N blocks
    add<<<N,1>>>(d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(&c, dc, size, hipMemcpyDeviceToHost);
    
    free(a); free(b); free(c);
    hipFree(da); hipFree(db); hipFree(dc);
    return 0;
}
