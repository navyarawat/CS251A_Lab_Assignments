#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

#define NUM 32

void CUDA_ERROR_EXIT(char str[]) {
    hipError_t err = hipGetLastError();
    if( err != hipSuccess){
        char temp[] = hipGetErrorString(err);
        printf("Cuda Error: '%s' for %s\n", temp, str);
        exit(-1);
    }
}

__global__ void D_Mul(int *dA, int *dB, int *dC) {
      int i = threadIdx.x;
      //int i = blockIdx.x * blockDim.x + threadIdx.x;
      dC[i] = dA[i] * dB[i];
}

int main (int argc, char **argv) {
    int size = NUM * sizeof(int);
    int* hA = (int *) malloc(size);
    int* hB = (int *) malloc(size);
    int* hC = (int *) malloc(size);

    if (!hA || !hB || !hC) {
        perror("malloc");
        exit(-1);
    }

    for(int ctr = 0; ctr < NUM; ++ctr)
        hA[ctr] = hB[ctr] = ctr + 1;        //apparently, this works

    //Allocate memory on the device (GPU)
    int *dA, *dB, *dC;
    hipMalloc(&dA,  size);
    CUDA_ERROR_EXIT("hipMalloc");
    hipMalloc(&dB,  size);
    CUDA_ERROR_EXIT("hipMalloc");
    hipMalloc(&dC,  size);
    CUDA_ERROR_EXIT("hipMalloc");

    //Copy hA --> dA and hB --> dB
    hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);
    CUDA_ERROR_EXIT("memcpy1");
    hipMemcpy(dB, hB, size, hipMemcpyHostToDevice);
    CUDA_ERROR_EXIT("memcpy1");

    //Invoke the kernel
    D_Mul<<<1, NUM>>>(dA, dB, dC);
    //int blocks = (NUM + 1023) >> 10;
    //D_Mul<<<blocks, 1024>>>(dA, dB, dC);
    CUDA_ERROR_EXIT("kernel invocation");
    printf("kernel successful\n");

    //Copy back results
    hipMemcpy(hC, dC, size, hipMemcpyDeviceToHost);
    CUDA_ERROR_EXIT("memcpy");
    for(int ctr = 0; ctr < NUM; ++ctr)
        printf("%d %d %d\n", hA[ctr], hB[ctr], hC[ctr]);

    free(hA); free(hB); free(hC);
    hipFree(dA); hipFree(dB); hipFree(dC);
    return 0;
}
